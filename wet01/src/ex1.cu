#include "hip/hip_runtime.h"
#include "ex1.h"

__device__
void prefix_sum(int arr[], int arr_size) 
{
    // int tid = threadIdx.x; 
    // int increment;
    // // TODO: check if blockdim.x should be arr_size
    // for (int stride = 1 ; stride < blockDim.x ; stride *= 2)
    // {
    //     if (tid >= stride)
    //     {
    //         increment = arr[tid - stride];
    //     }
    //     __syncthreads();
    //     if (tid >= stride)
    //     {
    //         arr[tid] += increment;
    //     }
    //     __syncthreads();
    // }
    for (int i = 1 ; i < arr_size ; i++)
    {
        arr[i] += arr[i-1];
    }
}

__device__
void test_maps(int* cdf, uchar* maps, int tile_x, int tile_y)
{
    int maps_start_index = ((tile_x * TILE_COUNT) + tile_y) * COLOR_COUNT;
    int i = 0;    
    
    for (i = 0 ; i < COLOR_COUNT ; i++)
    {
        maps[maps_start_index + i] = cdf[i] * (COLOR_COUNT - 1) / (TILE_WIDTH * TILE_WIDTH);
        if (tile_x == 5 && tile_y == 3)
        {
            printf("cdf[%d]: %d maps: %d\n", i, cdf[i], maps[maps_start_index + i]);
        }
    }
}


__global__
void calc_maps(int* cdf, uchar* maps, int tile_x, int tile_y)
{   
    // todo: check this
    int index = tile_x * TILE_COUNT + tile_y;
    int tid_x = threadIdx.x;    

    //printf("CDF: %d, maps: %d\n", cdf[tid_x], maps[index * COLOR_COUNT + tid_x]);
    maps[index * COLOR_COUNT + tid_x] = cdf[tid_x] * (COLOR_COUNT - 1) / (TILE_WIDTH * TILE_WIDTH);
    //printf("CDF: %d, maps: %d\n", cdf[tid_x], maps[index * COLOR_COUNT + tid_x]);
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__ 
void interpolate_device(uchar* maps , uchar *in_img, uchar* out_img);




// Note: our addition. Check if global is OK
__global__ void calc_histogram(int* hist, uchar* all_in, int tile_tid_x, int tile_tid_y)
{
    int pixels_per_threads = 4;
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    int row = (tile_tid_x * TILE_WIDTH) + tid_x; 
    int col = (tile_tid_y * TILE_WIDTH) + tid_y * pixels_per_threads;

    int index =  row * IMG_WIDTH + col; 

    int color_value = 0;
    for (int i = 0 ; i < pixels_per_threads ; i++)
    {
        color_value = all_in[index];
        atomicAdd(&hist[color_value], 1);
    }
    __syncthreads();
}


__device__
void test_histogram(int* hist, uchar* all_in, int tile_tid_x, int tile_tid_y)
{
    int color_value = 0;
    int index = 0;
    int x_index = 0;
    int y_index = 0;

    for (int i = 0 ; i < TILE_WIDTH ; i++)
    {
        for(int j = 0 ; j < TILE_WIDTH ; j++)
        {
            x_index = TILE_WIDTH * tile_tid_x + i;
            y_index = TILE_WIDTH * tile_tid_y + j;
            index = x_index * IMG_WIDTH + y_index;

            color_value = all_in[index];
            atomicAdd(&hist[color_value], 1);
        }
    }   
}

__global__ void process_image_kernel(uchar *all_in, uchar *all_out, uchar *maps) 
{
    int tile_tid_x = threadIdx.x;
    int tile_tid_y = threadIdx.y;

    int* hist = (int*) malloc(COLOR_COUNT * sizeof(int));
    memset(hist, 0, COLOR_COUNT * sizeof(int));

    // dim3 threads_count(TILE_WIDTH, TILE_HALF_WIDTH / 2);
    // calc_histogram<<<1, threads_count>>>(hist, all_in, tile_tid_x, tile_tid_y);
    test_histogram(hist, all_in, tile_tid_x, tile_tid_y);
    if (tile_tid_x == 5 && tile_tid_y == 3)
    {
        for (int i = 0 ; i < 256 ; i++)
        {
            printf("histogram[%d]: %d\n", i, hist[i]);
        }
    }
    
    __syncthreads();
  
    prefix_sum(hist, COLOR_COUNT); 

    if (tile_tid_x == 5 && tile_tid_y == 3)
    {
        for (int i = 0 ; i < 256 ; i++)
        {
            printf("prefix sum[%d]: %d\n", i, hist[i]);
        }
    }

    int maps_start_index = tile_tid_x * TILE_COUNT + tile_tid_y;
    
    // calc_maps<<<1, COLOR_COUNT>>>(hist, maps, tile_tid_x, tile_tid_y);
    test_maps(hist, maps, tile_tid_x, tile_tid_y);
    if (tile_tid_x == 5 && tile_tid_y == 3)
    {
        for (int i = 0 ; i < 256 ; i++)
        {
            printf("maps[%d]: %d\n", i, maps[maps_start_index + i]);
        }
    }

    free(hist);

    __syncthreads();

    if (tile_tid_x == 0 && tile_tid_y == 0)
    {

        for (int i = 0 ; i < 256 ; i++)
        {
            //printf("Histogram[%d]: %d\n", i, hist[i]);
        }
        interpolate_device(maps, all_in, all_out);
        for (int j = 0 ; j < 8 * 8 * 256 ; j++)
        {
            printf("maps[%d] = %d\n", j, maps[j]);
        }
        for (int i = 0 ; i < 1 * IMG_WIDTH * IMG_HEIGHT ; i++)
        {
            //printf("img_in: %d\n img_out: %d maps: %d\n", all_in[i], all_out[i], maps[i % 256]);
        }
    }

    return; 
}

/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context 
{
    uchar* in_img;
    uchar* out_img;
    uchar* maps;
    // TODO define task serial memory buffers
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;

    hipMalloc(&context->in_img, IMG_HEIGHT * IMG_WIDTH * sizeof(uchar));
    hipMalloc(&context->out_img, IMG_HEIGHT * IMG_WIDTH * sizeof(uchar));
    hipMalloc(&context->maps, TILE_COUNT * TILE_COUNT * COLOR_COUNT * sizeof(uchar));
    //TODO: allocate GPU memory for a single input image, a single output image, and maps

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out)
{
    // uchar test_image_in[512][512];
    // for (int j = 0 ; j < 512 ; j++)
    // {
    //     for (int k = 0 ; k < 512 ; k++)
    //     {
    //         test_image_in[j][k] = k % 64;
    //     }
    // }
    //for (int i = 0 ; i < N_IMAGES ; i++)
    for (int i = 0 ; i < 1 ; i++)
    {
        uchar* cur_images_in = &images_in[i * IMG_WIDTH * IMG_HEIGHT];
        uchar* cur_images_out = &images_out[i * IMG_WIDTH * IMG_HEIGHT];
        // TODO: restore this line
        hipMemcpy(context->in_img, cur_images_in, IMG_HEIGHT * IMG_WIDTH * sizeof(uchar), hipMemcpyHostToDevice);
        //hipMemcpy(context->in_img, test_image_in, IMG_HEIGHT * IMG_WIDTH * sizeof(uchar), hipMemcpyHostToDevice);

        // add GPU kernel invokation here
        dim3 tiles_count(TILE_COUNT, TILE_COUNT, 1);
        process_image_kernel<<<1, tiles_count>>>(context->in_img, context->out_img, context->maps);

        hipMemcpy(cur_images_out, context->out_img, IMG_HEIGHT * IMG_WIDTH * sizeof(uchar), hipMemcpyDeviceToHost);
    }
    //TODO: in a for loop:
    //   1. copy the relevant image from images_in to the GPU memory you allocated
    //   2. invoke GPU kernel on this image
    //   3. copy output from GPU memory to relevant location in images_out_gpu_serial
}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    //TODO: free resources allocated in task_serial_init
    hipFree(context->in_img);
    hipFree(context->out_img);
    hipFree(context->maps);

    free(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context 
{
    // TODO define bulk-GPU memory buffers
};

/* Allocate GPU memory for all the input images, output images, and maps.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    //TODO: allocate GPU memory for all the input images, output images, and maps

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
    //TODO: copy all input images from images_in to the GPU memory you allocated
    //TODO: invoke a kernel with N_IMAGES threadblocks, each working on a different image
    //TODO: copy output images from GPU memory to images_out
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    //TODO: free resources allocated in gpu_bulk_init

    free(context);
}
